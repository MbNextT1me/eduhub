﻿
#include "hip/hip_runtime.h"


#include <iostream>
#include <stdio.h>

#define BLOCK_SIZE 8
#define N 10000

/*
* Блок 1    Время   0.082880    Нитей 256
* Блок 8    Время   0.087264
* Блок 16   Время   0.095488
* Блок 32   Время   0.138336
* Блок 64   Время   0.284768
* Блок 128  Время   0.991424
* 
* Блок 128  Время   0.166720    Нитей 16
* Блок 64   Время   0.103435
* Блок 16   Время   0.107431
* Блок 1    Время   0.078080
* 
* Блок 1    Время   0.083536    Нитей 64
* Блок 8    Время   0.069653                
* Блок 16   Время   0.079514    
* Блок 64   Время   0.139431    
* 
* В целом заметно, что небольшие блоковые структуры достаточно сильно снижают затраты по времени, однако это
* также зависит и от количества тредов в самом блоке, поэтому перебор логично совершать по двум критериям,
* если интересует оптимальное время
* 
*/

__global__ void subKernel(int *a, int *b, int cols)
{
    int idx = cols * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    a[idx] -= b[idx];
}

int main()
{
    int** a = new int* [N];
    int** b = new int* [N];

    srand(NULL);

    for (size_t i = 0; i < N; i++)
    {
        a[i] = new int[N];
        b[i] = new int[N];
    }

    for (size_t i = 0; i < N; i++)
    {
        for (size_t j = 0; j < N; j++)
        {
            a[i][j] = rand() % 100;
            b[i][j] = rand() % 100;
        }
    }

    int* dev_a;
    int* dev_b;

    hipMalloc((void**)&dev_a, N * N * sizeof(int));
    hipMalloc((void**)&dev_b, N * N * sizeof(int));

    dim3 threadsInBlock = dim3(8, 4);
    dim3 blocksInGrid = dim3(BLOCK_SIZE, BLOCK_SIZE);

    int* a_line = new int[N * N];
    int* b_line = new int[N * N];

    for (size_t i = 0; i < N; i++)
    {
        for (size_t j = 0; j < N; j++)
        {
            a_line[i * N + j] = a[i][j];
            b_line[i * N + j] = b[i][j];
        }
    }

    /*for (size_t i = 0; i < 10; i++)
    {
        for (size_t j = 0; j < 10; j++)
        {
            printf("%d\t", a[i][j]);
        }
        printf("\n");
    }*/
    
    hipMemcpy(dev_a, a_line, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b_line, N * N * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventRecord(start, 0);

    subKernel <<< blocksInGrid, threadsInBlock >>> (dev_a, dev_b, N);

    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipMemcpy(a_line, dev_a, N * N *sizeof(int), hipMemcpyDeviceToHost);

    

    hipDeviceReset();

    for (size_t i = 0; i < 10; i++)
    {
        for (size_t j = 0; j < 10; j++)
        {
            printf("%d\t", a_line[i * N + j]);
        }
        printf("\n");
    }
    printf("Time in millis: %f", time);
}